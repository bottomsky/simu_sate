// Copyright 2024 The Trae Authors.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/**
 * @file j2_cuda_kernels.cu
 * @brief 包含用于J2摄动轨道外推和位置计算的CUDA内核和接口函数。
 *
 * 该文件为支持CUDA的GPU提供了J2摄动模型的核心计算逻辑。
 * 它定义了两个主要的CUDA内核：
 * 1. `j2_propagate_kernel`: 并行更新大量卫星的长期轨道要素（升交点赤经、近地点幅角、平近点角）。
 * 2. `compute_positions_kernel`: 将轨道要素转换为惯性系中的笛卡尔坐标。
 *
 * 文件还提供了C风格的外部接口函数，以便从C++代码中调用这些CUDA内核。
 * 如果编译时未启用CUDA (HAVE_CUDA_TOOLKIT 未定义)，则会提供这些接口的空实现，并打印警告信息。
 *
 * 数据布局：
 * 为了在GPU上实现高效的内存访问（合并访问），轨道要素和位置数据采用“结构数组”(SoA)的布局方式。
 * 例如，一个包含N个卫星的数组`elements`在内存中布局如下：
 * [a_1, a_2, ..., a_N, e_1, e_2, ..., e_N, i_1, ..., M_N]
 * 同样，`positions`数组布局为：
 * [x_1, x_2, ..., x_N, y_1, y_2, ..., y_N, z_1, ..., z_N]
 */

#ifdef HAVE_CUDA_TOOLKIT
#include <cuda_runtime.h>
#include <device_launch_parameters.h>
#endif

#include <hip/hip_runtime.h>
#include <cmath>
#include <cstddef>

// 当没有CUDA时，提供空的实现，以确保代码可以链接和编译。
#ifndef HAVE_CUDA_TOOLKIT
#include <iostream>
extern "C" {
/**
 * @brief J2轨道外推的CUDA接口（空实现）。
 * @param elements 指向轨道要素数据数组的指针（SoA布局）。
 * @param num_satellites 卫星数量。
 * @param dt 时间步长（秒）。
 * @param mu 地球引力常数。
 * @param re 地球赤道半径。
 * @param j2 地球J2摄动系数。
 */
void cuda_propagate_j2(double* elements, size_t num_satellites, double dt,
                      double mu, double re, double j2) {
    // 如果在没有CUDA支持的情况下调用此函数，则向标准错误流打印警告。
    std::cerr << "Warning: CUDA not available. Please use CPU_SCALAR or CPU_SIMD mode." << std::endl;
}

/**
 * @brief 从轨道要素计算位置的CUDA接口（空实现）。
 * @param elements 指向轨道要素数据数组的指针（SoA布局）。
 * @param positions 指向存储计算出的位置坐标的数组指针（SoA布局）。
 * @param num_satellites 卫星数量。
 */
void cuda_compute_positions(double* elements, double* positions, 
                           size_t num_satellites) {
    // 如果在没有CUDA支持的情况下调用此函数，则向标准错误流打印警告。
    std::cerr << "Warning: CUDA not available. Please use CPU_SCALAR or CPU_SIMD mode." << std::endl;
}

/**
 * @brief 优化的CUDA接口（空实现）：使用持久化缓冲区的J2外推
 */
void cuda_propagate_j2_persistent(double* d_a, double* d_e, double* d_i,
                                 double* d_O, double* d_w, double* d_M,
                                 size_t num_satellites, double dt,
                                 double mu, double re, double j2,
                                 void* stream) {
    std::cerr << "Warning: CUDA not available. Please use CPU_SCALAR or CPU_SIMD mode." << std::endl;
}

/**
 * @brief 优化的CUDA接口（空实现）：使用持久化缓冲区的位置计算
 */
void cuda_compute_positions_persistent(double* d_a, double* d_e, double* d_i,
                                      double* d_O, double* d_w, double* d_M,
                                      double* d_x, double* d_y, double* d_z,
                                      size_t num_satellites, void* stream) {
    std::cerr << "Warning: CUDA not available. Please use CPU_SCALAR or CPU_SIMD mode." << std::endl;
}
}
#else

// 使用__constant__内存来存储全局物理参数。
// 这允许GPU上的所有线程高效地访问这些只读值。
__constant__ double d_MU = 3.986004418e14; ///< 地球引力常数 (m^3/s^2)
__constant__ double d_RE = 6378137.0;     ///< 地球赤道半径 (m)
__constant__ double d_J2 = 1.08263e-3;     ///< 地球J2摄动系数

/**
 * @brief 将角度归一化到 [0, 2*PI) 范围内。
 * @param angle 要归一化的角度（弧度）。
 * @return 归一化后的角度（弧度）。
 */
__device__ double normalize_angle_cuda(double angle) {
    const double TWO_PI = 6.28318530717958647692528676655900576;
    angle = fmod(angle, TWO_PI);
    if (angle < 0) {
        angle += TWO_PI;
    }
    return angle;
}

/**
 * @brief J2摄动外推的CUDA内核。
 *
 * 该内核为每个卫星启动一个线程，并行计算由于J2摄动引起的轨道要素的长期变化。
 * 它只更新升交点赤经(O)、近地点幅角(w)和平近点角(M)，因为半长轴、偏心率和倾角在J2模型下是长期不变的。
 *
 * @param a 指向半长轴数组的设备指针。
 * @param e 指向偏心率数组的设备指针。
 * @param i 指向倾角数组的设备指针。
 * @param O 指向升交点赤经数组的设备指针（输入/输出）。
 * @param w 指向近地点幅角数组的设备指针（输入/输出）。
 * @param M 指向平近点角数组的设备指针（输入/输出）。
 * @param num_satellites 卫星总数。
 * @param dt 时间步长（秒）。
 */
__global__ void j2_propagate_kernel(double* a, double* e, double* i, 
                                   double* O, double* w, double* M,
                                   int num_satellites, double dt) {
    // 计算当前线程处理的卫星索引。
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_satellites) {
        // 从全局内存加载当前卫星的轨道要素到寄存器，以加快访问速度。
        double a_val = a[idx];
        double e_val = e[idx];
        double i_val = i[idx];
        double O_val = O[idx];
        double w_val = w[idx];
        double M_val = M[idx];
        
        // 计算平均角速度 n = sqrt(d_MU / a^3)
        double n = sqrt(d_MU / (a_val * a_val * a_val));
        
        // 计算J2摄动引起的长期变化率的公共因子，与CPU实现保持一致：
        // p = a * (1 - e^2)
        double one_minus_e2 = 1.0 - e_val * e_val;
        double p = a_val * one_minus_e2;
        // factor = (3/2) * J2 * n * (RE/p)^2
        double factor = 1.5 * d_J2 * n * (d_RE / p) * (d_RE / p);
        
        // 预计算三角函数值。
        double cos_i = cos(i_val);
        double sin_i = sin(i_val);
        double cos2_i = cos_i * cos_i;
        double sin2_i = sin_i * sin_i;
        
        // 计算升交点赤经、近地点幅角和平近点角的导数，并乘以时间步长得到变化量。
        // dO/dt = - (3/2) * n * J2 * (Re/p)^2 * cos(i)
        double dO = -factor * cos_i * dt;
        // dw/dt = (3/2) * n * J2 * (Re/p)^2 * (2 - 2.5 * sin^2(i))
        double dw = factor * (2.0 - 2.5 * sin2_i) * dt;
        // dM/dt 与CPU实现严格一致：n - factor * sqrt(1-e^2) * (1.5 * sin^2(i) - 0.5)
        double dM = (n - factor * sqrt(one_minus_e2) * (1.5 * sin2_i - 0.5)) * dt;
        
        // 更新轨道要素并将角度归一化到 [0, 2*PI) 范围。
        O[idx] = normalize_angle_cuda(O_val + dO);
        w[idx] = normalize_angle_cuda(w_val + dw);
        M[idx] = normalize_angle_cuda(M_val + dM);
    }
}

/**
 * @brief 将轨道要素转换为笛卡尔坐标的CUDA内核。
 *
 * 该内核为每个卫星启动一个线程，并行地将其轨道要素转换为地心惯性系(ECI)中的位置坐标(x, y, z)。
 *
 * @param a 指向半长轴数组的设备指针。
 * @param e 指向偏心率数组的设备指针。
 * @param i 指向倾角数组的设备指针。
 * @param O 指向升交点赤经数组的设备指针。
 * @param w 指向近地点幅角数组的设备指针。
 * @param M 指向平近点角数组的设备指针。
 * @param pos_x 指向x坐标数组的设备指针（输出）。
 * @param pos_y 指向y坐标数组的设备指针（输出）。
 * @param pos_z 指向z坐标数组的设备指针（输出）。
 * @param num_satellites 卫星总数。
 */
__global__ void compute_positions_kernel(double* a, double* e, double* i,
                                        double* O, double* w, double* M,
                                        double* pos_x, double* pos_y, double* pos_z,
                                        int num_satellites) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_satellites) {
        // 从全局内存加载轨道要素。
        double a_val = a[idx];
        double e_val = e[idx];
        double i_val = i[idx];
        double O_val = O[idx];
        double w_val = w[idx];
        double M_val = M[idx];
        
        // 求解开普勒方程 M = E - e*sin(E) 来计算偏近点角(E)。
        // 这里使用牛顿法进行一次迭代，对于小偏心率轨道足够精确。
        double E = M_val;  // 初始猜测
        E = E - (E - e_val * sin(E) - M_val) / (1.0 - e_val * cos(E));
        
        // 计算真近点角(nu)。
        double tan_nu_2 = sqrt((1.0 + e_val) / (1.0 - e_val)) * tan(E / 2.0);
        double nu = 2.0 * atan(tan_nu_2);
        
        // 计算卫星到地心的距离(r)。
        double r = a_val * (1.0 - e_val * cos(E));
        
        // 在轨道平面（周航坐标系）内计算位置。
        double x_perifocal = r * cos(nu);
        double y_perifocal = r * sin(nu);
        
        // 预计算从周航坐标系到地心惯性系的旋转矩阵所需的三角函数。
        double cosO = cos(O_val);
        double sinO = sin(O_val);
        double cosi = cos(i_val);
        double sini = sin(i_val);
        double cosw = cos(w_val);
        double sinw = sin(w_val);
        
        // 执行坐标旋转，将位置从周航坐标系转换到地心惯性系(ECI)。
        pos_x[idx] = (cosO*cosw - sinO*sinw*cosi) * x_perifocal + 
                     (-cosO*sinw - sinO*cosw*cosi) * y_perifocal;
        pos_y[idx] = (sinO*cosw + cosO*sinw*cosi) * x_perifocal + 
                     (-sinO*sinw + cosO*cosw*cosi) * y_perifocal;
        pos_z[idx] = sinw*sini * x_perifocal + cosw*sini * y_perifocal;
    }
}

// C接口函数，封装CUDA内核调用，以便从C++代码中调用。
extern "C" {
    /**
     * @brief J2轨道外推的CUDA接口函数。
     *
     * 该函数负责管理内存传输（主机到设备，设备到主机）和启动`j2_propagate_kernel`内核。
     *
     * @param elements 指向主机内存中轨道要素数据的指针（SoA布局）。
     * @param num_satellites 卫星数量。
     * @param dt 时间步长（秒）。
     * @param mu 地球引力常数。
     * @param re 地球赤道半径。
     * @param j2 地球J2摄动系数。
     */
    void cuda_propagate_j2(double* elements, size_t num_satellites, double dt, 
                          double mu, double re, double j2) {
        
        // 将物理常数从主机内存复制到设备的__constant__内存。
        cudaMemcpyToSymbol(d_MU, &mu, sizeof(double));
        cudaMemcpyToSymbol(d_RE, &re, sizeof(double));
        cudaMemcpyToSymbol(d_J2, &j2, sizeof(double));
        
        // 在GPU设备上为每个轨道要素数组分配内存。
        double *d_a, *d_e, *d_i, *d_O, *d_w, *d_M;
        size_t size = num_satellites * sizeof(double);
        
        cudaMalloc(&d_a, size);
        cudaMalloc(&d_e, size);
        cudaMalloc(&d_i, size);
        cudaMalloc(&d_O, size);
        cudaMalloc(&d_w, size);
        cudaMalloc(&d_M, size);
        
        // 将数据从主机内存复制到设备内存。
        double* a = elements;
        double* e = elements + num_satellites;
        double* i = elements + 2 * num_satellites;
        double* O = elements + 3 * num_satellites;
        double* w = elements + 4 * num_satellites;
        double* M = elements + 5 * num_satellites;
        
        cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_e, e, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_i, i, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_O, O, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_w, w, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_M, M, size, cudaMemcpyHostToDevice);
        
        // 启动内核。每个线程处理一个卫星。
        int threadsPerBlock = 256;
        int blocksPerGrid = (static_cast<int>(num_satellites) + threadsPerBlock - 1) / threadsPerBlock;
        j2_propagate_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_e, d_i, d_O, d_w, d_M, static_cast<int>(num_satellites), dt);
        cudaDeviceSynchronize();
        
        // 将结果从设备内存复制回主机内存。
        cudaMemcpy(O, d_O, size, cudaMemcpyDeviceToHost);
        cudaMemcpy(w, d_w, size, cudaMemcpyDeviceToHost);
        cudaMemcpy(M, d_M, size, cudaMemcpyDeviceToHost);
        
        // 释放设备内存。
        cudaFree(d_a);
        cudaFree(d_e);
        cudaFree(d_i);
        cudaFree(d_O);
        cudaFree(d_w);
        cudaFree(d_M);
    }

    /**
     * @brief 计算位置的CUDA接口函数。
     *
     * 该函数负责内存管理和启动`compute_positions_kernel`内核，将轨道要素转换为ECI坐标。
     *
     * @param elements 指向主机内存中轨道要素数据的指针（SoA布局）。
     * @param positions 指向主机内存中输出位置数据的指针（SoA布局）。
     * @param num_satellites 卫星数量。
     */
    void cuda_compute_positions(double* elements, double* positions, size_t num_satellites) {
        // 在GPU设备上为轨道要素和位置数组分配内存。
        double *d_a, *d_e, *d_i, *d_O, *d_w, *d_M;
        double *d_x, *d_y, *d_z;
        size_t size = num_satellites * sizeof(double);
        
        cudaMalloc(&d_a, size);
        cudaMalloc(&d_e, size);
        cudaMalloc(&d_i, size);
        cudaMalloc(&d_O, size);
        cudaMalloc(&d_w, size);
        cudaMalloc(&d_M, size);
        cudaMalloc(&d_x, size);
        cudaMalloc(&d_y, size);
        cudaMalloc(&d_z, size);
        
        // 将数据从主机内存复制到设备内存。
        double* a = elements;
        double* e = elements + num_satellites;
        double* i = elements + 2 * num_satellites;
        double* O = elements + 3 * num_satellites;
        double* w = elements + 4 * num_satellites;
        double* M = elements + 5 * num_satellites;
        
        cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_e, e, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_i, i, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_O, O, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_w, w, size, cudaMemcpyHostToDevice);
        cudaMemcpy(d_M, M, size, cudaMemcpyHostToDevice);
        
        // 启动内核。
        int threadsPerBlock = 256;
        int blocksPerGrid = (static_cast<int>(num_satellites) + threadsPerBlock - 1) / threadsPerBlock;
        compute_positions_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_a, d_e, d_i, d_O, d_w, d_M, d_x, d_y, d_z, static_cast<int>(num_satellites));
        cudaDeviceSynchronize();
        
        // 将结果从设备内存复制回主机内存。
        double* x = positions;
        double* y = positions + num_satellites;
        double* z = positions + 2 * num_satellites;
        
        cudaMemcpy(x, d_x, size, cudaMemcpyDeviceToHost);
        cudaMemcpy(y, d_y, size, cudaMemcpyDeviceToHost);
        cudaMemcpy(z, d_z, size, cudaMemcpyDeviceToHost);
        
        // 释放设备内存。
        cudaFree(d_a);
        cudaFree(d_e);
        cudaFree(d_i);
        cudaFree(d_O);
        cudaFree(d_w);
        cudaFree(d_M);
        cudaFree(d_x);
        cudaFree(d_y);
        cudaFree(d_z);
    }
    
    /**
     * @brief 优化的CUDA接口：使用持久化缓冲区的J2外推
     * 
     * @param d_a 设备端半长轴数组指针
     * @param d_e 设备端偏心率数组指针  
     * @param d_i 设备端倾角数组指针
     * @param d_O 设备端升交点赤经数组指针
     * @param d_w 设备端近地点幅角数组指针
     * @param d_M 设备端平近点角数组指针
     * @param num_satellites 卫星数量
     * @param dt 时间步长
     * @param mu 地球引力常数
     * @param re 地球赤道半径 
     * @param j2 地球J2摄动系数
     * @param stream CUDA流（异步执行）
     */
    void cuda_propagate_j2_persistent(double* d_a, double* d_e, double* d_i,
                                     double* d_O, double* d_w, double* d_M,
                                     size_t num_satellites, double dt,
                                     double mu, double re, double j2,
                                     cudaStream_t stream) {
        // 更新常量内存
        cudaMemcpyToSymbol(d_MU, &mu, sizeof(double));
        cudaMemcpyToSymbol(d_RE, &re, sizeof(double));
        cudaMemcpyToSymbol(d_J2, &j2, sizeof(double));
        
        // 启动内核（异步）
        int threadsPerBlock = 256;
        int blocksPerGrid = (static_cast<int>(num_satellites) + threadsPerBlock - 1) / threadsPerBlock;
        j2_propagate_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            d_a, d_e, d_i, d_O, d_w, d_M, static_cast<int>(num_satellites), dt);
    }
    
    /**
     * @brief 优化的CUDA接口：使用持久化缓冲区的位置计算
     */
    void cuda_compute_positions_persistent(double* d_a, double* d_e, double* d_i,
                                          double* d_O, double* d_w, double* d_M,
                                          double* d_x, double* d_y, double* d_z,
                                          size_t num_satellites, cudaStream_t stream) {
        // 启动内核（异步）
        int threadsPerBlock = 256;
        int blocksPerGrid = (static_cast<int>(num_satellites) + threadsPerBlock - 1) / threadsPerBlock;
        compute_positions_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            d_a, d_e, d_i, d_O, d_w, d_M, d_x, d_y, d_z, static_cast<int>(num_satellites));
    }
}
#endif // HAVE_CUDA_TOOLKIT